#include "hip/hip_runtime.h"
#include "CudaFlow.h"

///search image
texture<float, 2, hipReadModeElementType> texKernelImage;
texture<float, 2, hipReadModeElementType> texSearchImage;


__global__ void Correlation1x1Kernel(float kernel, float*searchSpace, float*output,
	int width, int height, int stride) {
	const int ix = threadIdx.x + blockIdx.x * blockDim.x;
	const int iy = threadIdx.y + blockIdx.y * blockDim.y;

	const int pos = ix + iy * stride;

	if (ix >= width || iy >= height) return;

	float diff = searchSpace[iy*stride + ix] - kernel;
	output[pos] = diff*diff;
}

__global__ void CorrelationKernel(float* kernel, float* searchSpace, float* output,
	int width, int height, int stride, int kernelSize) {

	const int ix = threadIdx.x + blockIdx.x * blockDim.x;
	const int iy = threadIdx.y + blockIdx.y * blockDim.y;

	const int pos = ix + iy * stride;

	if (ix >= width || iy >= height) return;

	float total = 0.0f;
	int offset = (kernelSize - 1) / 2;
	for (int j = 0; j < kernelSize; j++) {
		for (int i = 0; i < kernelSize; i++) {
			int col = ix + i - offset;
			int row = iy + j - offset;
			//correlate
			float diff = searchSpace[row*stride + col] - kernel[j*kernelSize + i];
			total += diff*diff;
			//total += abs(searchSpace[row*stride + col] * kernel[j*kernelSize + i]);
		}
	}
	//normalize
	output[pos] = total;// / (kernelSize*kernelSize);
}

__global__ void CorrelationKernelSamplingKernel(int x, int y, float* kernel, int width, int height, int kernelSize) {
	const int ix = threadIdx.x + blockIdx.x * blockDim.x;
	const int iy = threadIdx.y + blockIdx.y * blockDim.y;
	const int pos = ix + iy * kernelSize;
	if (ix >= kernelSize || iy >= kernelSize) return;

	float texx = ((float)ix + (float)x + 0.5f - (kernelSize - 1) / 2) / (float)width;
	float texy = ((float)iy + (float)y + 0.5f - (kernelSize - 1) / 2) / (float)height;

	kernel[pos] = tex2D(texKernelImage, texx, texy);
}

__global__ void CorrelationSearchSamplingKernel(int x, int y, float* searchSpace, int maxSearchWidth, int maxSearchHeight, int corrStride, int width, int height) {
	const int ix = threadIdx.x + blockIdx.x * blockDim.x;
	const int iy = threadIdx.y + blockIdx.y * blockDim.y;
	const int pos = ix + iy * corrStride;
	if (ix >= maxSearchWidth || iy >= maxSearchHeight) return;

	float texx = ((float)ix + (float)x + 0.5f - (maxSearchWidth) / 2) / (float)width;
	float texy = ((float)iy + (float)y + 0.5f - (maxSearchHeight) / 2) / (float)height;

	searchSpace[pos] = tex2D(texSearchImage, texx, texy);
}



void sor::CudaFlow::Correlation(float* kernel, float* searchSpace, float* output)
{
	dim3 threads(32, 12);
	dim3 blocks(iDivUp(corrMaxSearchWidth, threads.x), iDivUp(corrMaxSearchHeight, threads.y));

	CorrelationKernel << < blocks, threads >> > (kernel, searchSpace, output, corrMaxSearchWidth, corrMaxSearchHeight, corrStride, corrKernelSize);
}

void sor::CudaFlow::Correlation1x1(float kernel, float* searchSpace, float* output)
{
	dim3 threads(32, 12);
	dim3 blocks(iDivUp(corrMaxSearchWidth, threads.x), iDivUp(corrMaxSearchHeight, threads.y));

	Correlation1x1Kernel << < blocks, threads >> > (kernel, searchSpace, output, corrMaxSearchWidth, corrMaxSearchHeight, corrStride);
}

void sor::CudaFlow::CorrelationBindTextures(float* im0, float*im1, int w, int h, int s) {
	texSearchImage.addressMode[0] = hipAddressModeClamp;
	texSearchImage.addressMode[1] = hipAddressModeClamp;
	texSearchImage.filterMode = hipFilterModeLinear;
	texSearchImage.normalized = true;

	texKernelImage.addressMode[0] = hipAddressModeClamp;
	texKernelImage.addressMode[1] = hipAddressModeClamp;
	texKernelImage.filterMode = hipFilterModeLinear;
	texKernelImage.normalized = true;

	hipChannelFormatDesc desc = hipCreateChannelDesc<float>();

	checkCudaErrors(hipBindTexture2D(0, texKernelImage, im0, w, h, s * sizeof(float)));
	checkCudaErrors(hipBindTexture2D(0, texSearchImage, im1, w, h, s * sizeof(float)));
}

void sor::CudaFlow::CorrelationKernelSampling(int x, int y, float* kernel, int w, int h) {
	dim3 threads(corrKernelSize, corrKernelSize);
	dim3 blocks(iDivUp(corrKernelSize, threads.x), iDivUp(corrKernelSize, threads.y));
	CorrelationKernelSamplingKernel << < blocks, threads >> > (x, y, kernel, w, h, corrKernelSize);
}

void sor::CudaFlow::CorrelationSearchSampling(int x, int y, float* searchSpace) {
	dim3 threads(BlockWidth, BlockHeight);
	dim3 blocks(iDivUp(corrMaxSearchWidth, threads.x), iDivUp(corrMaxSearchHeight, threads.y));
	CorrelationSearchSamplingKernel << < blocks, threads >> > (x, y, searchSpace, corrMaxSearchWidth, corrMaxSearchHeight, corrStride, width, height);
}


__global__ void GetValueKernel(float *input, int idx, float &value) {
	value = input[idx];
}

void sor::CudaFlow::GetValue(float *input, int idx, float &value) {
	GetValueKernel << <1, 1 >> > (input, idx, value);
}