#include "hip/hip_runtime.h"
#include "CudaFlow.h"


__global__ void SolveSmoothDualTVGlobalKernel (float* duhat, float* dvhat,
	float* pu1, float* pu2,
	float* pv1, float* pv2,
	int width, int height, int stride,
	float tau, float theta,
	float *pu1s, float *pu2s,
	float *pv1s, float* pv2s)
{
	const int ix = threadIdx.x + blockIdx.x * blockDim.x;
	const int iy = threadIdx.y + blockIdx.y * blockDim.y;

	const int pos = ix + iy * stride;

	if (ix >= width || iy >= height) return;
	
	int left = (ix - 1) + iy * stride;
	int right = (ix + 1) + iy * stride;
	int down = ix + (iy - 1) * stride;
	int up = ix + (iy + 1) * stride;

	//solve derivatives of duhat and dvhat
	float dux, duy, dvx, dvy;
	if ((ix + 1) >= width) {
		//dux = duhat[pos] - duhat[left];
		//dvx = dvhat[pos] - dvhat[left];
		dux = 0;
		dvx = 0;
	}
	else {
		dux = duhat[right] - duhat[pos];
		dvx = dvhat[right] - dvhat[pos];
	}
	if ((iy + 1) >= height) {
		//duy = duhat[pos] - duhat[down];
		//dvy = dvhat[pos] - dvhat[down];
		duy = 0;
		dvy = 0;
	}
	else {
		duy = duhat[up] - duhat[pos];
		dvy = dvhat[up] - dvhat[pos];
	}
	float magdu = sqrt(dux*dux + duy*duy);
	float magdv = sqrt(dvx*dvx + dvy*dvy);
	float fac = tau / theta;

	float pu1sub = pu1[pos];
	float pu2sub = pu2[pos];
	float pv1sub = pv1[pos];
	float pv2sub = pv2[pos];

	for (int k = 0; k < 1; k++) {
		pu1sub = (pu1sub + fac*dux) / (1 + fac*magdu);
		pu2sub = (pu2sub + fac*duy) / (1 + fac*magdu);
		pv1sub = (pv1sub + fac*dvx) / (1 + fac*magdv);
		pv2sub = (pv2sub + fac*dvy) / (1 + fac*magdv);
	}
	pu1s[pos] = pu1sub;
	pu2s[pos] = pu2sub;
	pv1s[pos] = pv1sub;
	pv2s[pos] = pv2sub;
}

///////////////////////////////////////////////////////////////////////////////
/// \brief compute image derivatives
///
/// \param[in]  I0  source image
/// \param[in]  I1  tracked image
/// \param[in]  w   image width
/// \param[in]  h   image height
/// \param[in]  s   image stride
/// \param[out] Ix  x derivative
/// \param[out] Iy  y derivative
/// \param[out] Iz  temporal derivative
///////////////////////////////////////////////////////////////////////////////

void sor::CudaFlow::SolveSmoothDualTVGlobal(float *duhat, float *dvhat,
	float *pu1, float *pu2, float *pv1, float *pv2,
	int w, int h, int s,
	float tau, float theta,
	float *pu1s, float*pu2s,
	float *pv1s, float *pv2s
	)
{
	dim3 threads(BlockWidth, BlockHeight);
	dim3 blocks(iDivUp(w, threads.x), iDivUp(h, threads.y));

	SolveSmoothDualTVGlobalKernel <<< blocks, threads >>> (duhat, dvhat, 
		pu1, pu2, pv1, pv2, 
		w, h, s, 
		tau, theta,
		pu1s, pu2s, pv1s, pv2s);
}
