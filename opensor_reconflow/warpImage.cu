#include "hip/hip_runtime.h"
#include "CudaFlow.h"

/// image to warp
texture<float, 2, hipReadModeElementType> texToWarp;

///////////////////////////////////////////////////////////////////////////////
/// \brief warp image with a given displacement field, CUDA kernel.
/// \param[in]  width   image width
/// \param[in]  height  image height
/// \param[in]  stride  image stride
/// \param[in]  u       horizontal displacement
/// \param[in]  v       vertical displacement
/// \param[out] out     result
///////////////////////////////////////////////////////////////////////////////
__global__ void WarpingKernel(int width, int height, int stride,
	const float *u, const float *v, float *out)
{
	const int ix = threadIdx.x + blockIdx.x * blockDim.x;
	const int iy = threadIdx.y + blockIdx.y * blockDim.y;

	const int pos = ix + iy * stride;

	if (ix >= width || iy >= height) return;

	float x = ((float)ix + u[pos] + 0.5f) / (float)width;
	float y = ((float)iy + v[pos] + 0.5f) / (float)height;

	out[pos] = tex2D(texToWarp, x, y);
}

///////////////////////////////////////////////////////////////////////////////

void sor::CudaFlow::WarpImage(const float *src, int w, int h, int s,
	const float *u, const float *v, float *out)
{
	dim3 threads(BlockWidth, BlockHeight);
	dim3 blocks(iDivUp(w, threads.x), iDivUp(h, threads.y));

	// mirror if a coordinate value is out-of-range
	texToWarp.addressMode[0] = hipAddressModeMirror;
	texToWarp.addressMode[1] = hipAddressModeMirror;
	texToWarp.filterMode = hipFilterModeLinear;
	texToWarp.normalized = true;

	hipChannelFormatDesc desc = hipCreateChannelDesc<float>();

	hipBindTexture2D(0, texToWarp, src, w, h, s * sizeof(float));

	WarpingKernel << <blocks, threads >> >(w, h, s, u, v, out);
}
