#include "hip/hip_runtime.h"
#include "CudaFlow.h"


__global__
void SolveDataL1InpaintKernel(const float *duhat0, const float *dvhat0,
	const float *mask0, const float *mask1,
	const float *pu1, const float *pu2,
	const float *pv1, const float *pv2,
	const float *Ix, const float *Iy, const float *It,
	int width, int height, int stride,
	float lambda, float theta,
	float *duhat1, float *dvhat1)
{
	int ix = blockIdx.x * blockDim.x + threadIdx.x;
	int iy = blockIdx.y * blockDim.y + threadIdx.y;        // current row 

	float dix, diy, dit, duhat, dvhat, du, dv;

	if ((iy < height) && (ix < width))
	{
		int pos = ix + iy * stride;      // current pixel index
		dix = Ix[pos];
		diy = Iy[pos];
		dit = It[pos];
		float duhat = duhat0[pos];
		float dvhat = dvhat0[pos];
		float dmask0 = mask0[pos];

		//problem 1a
		float rho = (dix*duhat + diy*dvhat + dit);
		float upper = lambda*theta*(dix*dix + diy*diy);
		float lower = -lambda*theta*(dix*dix + diy*diy);;

		if ((rho <= upper) && (rho >= lower)) {
			float magi = dix*dix + diy*diy;
			if (magi != 0) {
				du = duhat - rho*dix / magi;
				dv = dvhat - rho*diy / magi;
			}
			else {
				du = duhat;
				dv = dvhat;
			}

		}
		else if (rho < lower) {
			du = duhat + lambda*theta*dix;
			dv = dvhat + lambda*theta*diy;
		}
		else if (rho > upper) {
			du = duhat - lambda*theta*dix;
			dv = dvhat - lambda*theta*diy;
		}

		if (dmask0 == 0.0f) {
			du = duhat;
			dv = dvhat;
		}

		//problem 1b
		float divpu, divpv;
		int left = (ix - 1) + iy * stride;
		int right = (ix + 1) + iy * stride;
		int down = ix + (iy - 1) * stride;
		int up = ix + (iy + 1) * stride;

		if ((ix - 1) < 0) {
			if ((iy - 1) < 0) {
				//divpu = pu1[right] - pu1[pos] + pu2[up] - pu2[pos];
				//divpv = pv1[right] - pv1[pos] + pv2[up] - pv2[pos];
				divpu = pu1[pos] + pu2[pos];
				divpv = pv1[pos] + pv2[pos];
			}
			else {
				//divpu = pu1[right] - pu1[pos] + pu2[pos] - pu2[down];
				//divpv = pv1[right] - pv1[pos] + pv2[pos] - pv2[down];
				divpu = pu1[pos] + pu2[pos] - pu2[down];
				divpv = pv1[pos] + pv2[pos] - pv2[down];
			}
		}
		else {
			if ((iy - 1) < 0) {
				//divpu = pu1[pos] - pu1[left] + pu2[up] - pu2[pos];
				//divpv = pv1[pos] - pv1[left] + pv2[up] - pv2[pos];
				divpu = pu1[pos] - pu1[left] + pu2[pos];
				divpv = pv1[pos] - pv1[left] + pv2[pos];
			}
			else {
				divpu = pu1[pos] - pu1[left] + pu2[pos] - pu2[down];
				divpv = pv1[pos] - pv1[left] + pv2[pos] - pv2[down];
			}
		}

		duhat1[pos] = du + theta*divpu;
		dvhat1[pos] = dv + theta*divpv;
	}

}


void sor::CudaFlow::SolveDataL1Inpaint(const float *duhat0, const float *dvhat0,
	const float *mask0, const float *mask1,
	const float *pu1, const float *pu2,
	const float *pv1, const float *pv2,
	const float *Ix, const float *Iy, const float *Iz,
	int w, int h, int s,
	float lambda, float theta,
	float *duhat1, float *dvhat1)
{
	// CTA size
	dim3 threads(BlockWidth, BlockHeight);
	// grid size
	dim3 blocks(iDivUp(w, threads.x), iDivUp(h, threads.y));

	SolveDataL1InpaintKernel << < blocks, threads >> > (duhat0, dvhat0,
		mask0, mask1,
		pu1, pu2,
		pv1, pv2,
		Ix, Iy, Iz,
		w, h, s,
		lambda, theta,
		duhat1, dvhat1);
}
