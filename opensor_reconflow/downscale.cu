#include "hip/hip_runtime.h"
#include "CudaFlow.h"

/// image to downscale
texture<float, 2, hipReadModeElementType> texFine;

__global__ void DownscaleKernel(int width, int height, int stride, float *out)
{
	const int ix = threadIdx.x + blockIdx.x * blockDim.x;
	const int iy = threadIdx.y + blockIdx.y * blockDim.y;

	if (ix >= width || iy >= height)
	{
		return;
	}

	float dx = 1.0f / (float)width;
	float dy = 1.0f / (float)height;

	float x = ((float)ix + 0.5f) * dx;
	float y = ((float)iy + 0.5f) * dy;

	int pos = ix + iy * stride;

	out[pos] = 0.25f * (tex2D(texFine, x - dx * 0.25f, y) + tex2D(texFine, x + dx * 0.25f, y) +
		tex2D(texFine, x, y - dy * 0.25f) + tex2D(texFine, x, y + dy * 0.25f));
}

__global__ void DownscaleScalingKernel(int width, int height, int stride, float scale, float *out)
{
	const int ix = threadIdx.x + blockIdx.x * blockDim.x;
	const int iy = threadIdx.y + blockIdx.y * blockDim.y;

	if (ix >= width || iy >= height)
	{
		return;
	}

	float dx = 1.0f / (float)width;
	float dy = 1.0f / (float)height;

	float x = ((float)ix + 0.5f) * dx;
	float y = ((float)iy + 0.5f) * dy;

	int pos = ix + iy * stride;

	out[pos] = scale * 0.25f * (tex2D(texFine, x - dx * 0.25f, y) + tex2D(texFine, x + dx * 0.25f, y) +
		tex2D(texFine, x, y - dy * 0.25f) + tex2D(texFine, x, y + dy * 0.25f));
}

void sor::CudaFlow::Downscale(const float *src, int width, int height, int stride,
	int newWidth, int newHeight, int newStride, float *out)
{
	dim3 threads(BlockWidth, BlockHeight);
	dim3 blocks(iDivUp(newWidth, threads.x), iDivUp(newHeight, threads.y));

	// mirror if a coordinate value is out-of-range
	texFine.addressMode[0] = hipAddressModeMirror;
	texFine.addressMode[1] = hipAddressModeMirror;
	texFine.filterMode = hipFilterModeLinear;
	texFine.normalized = true;

	hipChannelFormatDesc desc = hipCreateChannelDesc<float>();

	checkCudaErrors(hipBindTexture2D(0, texFine, src, width, height, stride * sizeof(float)));

	DownscaleKernel <<< blocks, threads >>>(newWidth, newHeight, newStride, out);
}

void sor::CudaFlow::Downscale(const float *src, int width, int height, int stride,
	int newWidth, int newHeight, int newStride, float scale, float *out)
{
	dim3 threads(BlockWidth, BlockHeight);
	dim3 blocks(iDivUp(newWidth, threads.x), iDivUp(newHeight, threads.y));

	// mirror if a coordinate value is out-of-range
	texFine.addressMode[0] = hipAddressModeMirror;
	texFine.addressMode[1] = hipAddressModeMirror;
	texFine.filterMode = hipFilterModeLinear;
	texFine.normalized = true;

	hipChannelFormatDesc desc = hipCreateChannelDesc<float>();

	checkCudaErrors(hipBindTexture2D(0, texFine, src, width, height, stride * sizeof(float)));

	DownscaleScalingKernel << < blocks, threads >> >(newWidth, newHeight, newStride, scale, out);
}