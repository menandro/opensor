#include "hip/hip_runtime.h"
#include "CudaFlow.h"

__global__
void AddKernel(const float *op1, const float *op2, int count, float *sum)
{
	const int pos = threadIdx.x + blockIdx.x * blockDim.x;

	if (pos >= count) return;

	sum[pos] = op1[pos] + op2[pos];
}

void sor::CudaFlow::Add(const float *op1, const float *op2, int count, float *sum)
{
	dim3 threads(256);
	dim3 blocks(iDivUp(count, threads.x));
	AddKernel <<< blocks, threads >>>(op1, op2, count, sum);
}
